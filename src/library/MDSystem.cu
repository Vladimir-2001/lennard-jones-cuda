#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <hip/hip_runtime_api.h>

#include <math.h>

// cube length
__device__ __constant__ float L;

// whether minimum image convention periodic boundary conditions apply
__device__ __constant__ float periodic;

// for the radial distribution function
__device__ __constant__ float dr2;

// pressure via the virial theorem
__device__ float pressure;

// tau_xy of stress-eneryg tensor via virial theorem
__device__ float Pxy;

//__device__ float4
__device__ void
bodyBodyInteraction(float4 &ai, float4 bi, float4 &bj, int *RDF)
{
  float3 r;

  r.x = bj.x - bi.x;
  r.y = bj.y - bi.y;
  r.z = bj.z - bi.z;

  // Minimum image convention
  r.x += -periodic * L * lround(r.x / L);
  r.y += -periodic * L * lround(r.y / L);
  r.z += -periodic * L * lround(r.z / L);

  float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;

  if (distSqr > 1e-6f)
  {
    int indr2 = min(__float2int_rd(distSqr / dr2), 255);

    if (RDF != NULL)
      atomicAdd(&RDF[indr2], 1);

    float invDist2 = 1.0f / distSqr;
    float invDist6 = invDist2 * invDist2 * invDist2;

    float s = invDist2 * (12.0f * invDist6 * invDist6 - 6.0f * invDist6);

    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;
    ai.w += invDist6 * invDist6 - invDist6;
    bj.w += s * distSqr;

    atomicAdd(&Pxy, -r.x * r.y * s);
  }
}


// This is the "tile_calculation" function from the GPUG3 article.
__device__ float4 tile_force(float4 &myPos, float4 accel, int maxnumb, int *RDF)
{
  extern __shared__ float4 sharedPos[];
  int i;

#pragma unroll 8
  for (i = 0; i < maxnumb; ++i)
  {
    bodyBodyInteraction(accel, sharedPos[i], myPos, RDF);
  }

  return accel;
}

__device__ float4
computeBodyAccel(float4 &bodyPos, float4* positions, int numBodies, int maxnumblast, int *RDF)
{
  extern __shared__ float4 sharedPos[];

  float4 frc = { 0.0f, 0.0f, 0.0f, 0.0f };

  int start = 0;
  int tile = 0;
  int finish = start + numBodies - blockDim.x;

  bodyPos.w = 0.f;

  for (int i = start; i < finish; i += blockDim.x, tile++)
  {
    sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

    __syncthreads();
    frc = tile_force(bodyPos, frc, blockDim.x, RDF);
    __syncthreads();
  }

  if (threadIdx.x < maxnumblast)
    sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

  __syncthreads();

  frc = tile_force(bodyPos, frc, maxnumblast, RDF);
  __syncthreads();

  // Lennard-Jones factor
  frc.x *= 4.f;
  frc.y *= 4.f;
  frc.z *= 4.f;
  bodyPos.w *= 4.0f / 2.0f / 3.0f;

  return frc;
}

__global__ void
calculateForces(float4* Pos, float4* Force,
  int numBodies, int calcmaxnumblast, int *RDF)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int g = blockIdx.x;
  int* gmem = RDF + g * 256;
  for (int i = threadIdx.x; i < 256; i += blockDim.x)
    gmem[i] = 0;
  __syncthreads();

  int index2 = index;
  if (index2 >= numBodies) {
    index2 = numBodies - 1;
    gmem = NULL;
  }

  float4 pos = Pos[index2];
  float4 accel = computeBodyAccel(pos, Pos, numBodies, calcmaxnumblast, gmem);

  if (index < numBodies) {
    Force[index2] = accel;
    atomicAdd(&pressure, pos.w);
  }
}

__global__ void
RDFmerger(int* RDF, int blocks, int *RDFout)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= 256)
    return;

  int total = 0;
  for (int j = 0; j < blocks; ++j) {
    total += RDF[index + blockDim.x * j];
  }

  RDFout[index] = total;
}

extern "C"
{

  void allocateNBodyArrays(float* vel[2], int numBodies)
  {
    // 4 floats each for alignment reasons
    unsigned int memSize = sizeof(float) * 4 * numBodies;

    checkCudaErrors(hipMalloc((void**)&vel[0], memSize));
    checkCudaErrors(hipMalloc((void**)&vel[1], memSize));
  }

  void allocateArray(float** dest, int number)
  {
    // 4 floats each for alignment reasons
    unsigned int memSize = sizeof(float) * 4 * number;

    checkCudaErrors(hipMalloc((void**)dest, memSize));
  }

  void deleteNBodyArrays(float* vel[2])
  {
    checkCudaErrors(hipFree((void**)vel[0]));
    checkCudaErrors(hipFree((void**)vel[1]));
  }

  void deleteArray(float* arr)
  {
    checkCudaErrors(hipFree((void**)arr));
  }

  void allocateIntArray(int** dest, int number)
  {
    // 4 floats each for alignment reasons
    unsigned int memSize = sizeof(int) * number;

    checkCudaErrors(hipMalloc((void**)dest, memSize));
  }

  void deleteIntArray(int* arr)
  {
    checkCudaErrors(hipFree((void**)arr));
  }

  void copyArrayFromDevice(float* host,
    const float* device,
    unsigned int pbo,
    int numBodies)
  {
    //if (pbo)
    //    checkCudaErrors(cudaGLMapBufferObject((void**)&device, pbo));
    checkCudaErrors(hipMemcpy(host, device, numBodies * 4 * sizeof(float),
      hipMemcpyDeviceToHost));
    //if (pbo)
    //   checkCudaErrors(cudaGLUnmapBufferObject(pbo));
  }

  void copyArrayToDevice(float* device, const float* host, int numBodies)
  {
    checkCudaErrors(hipMemcpy(device, host, numBodies * 4 * sizeof(float),
      hipMemcpyHostToDevice));
  }

  void registerGLBufferObject(unsigned int pbo)
  {
    //checkCudaErrors(cudaGLRegisterBufferObject(pbo));
  }

  void unregisterGLBufferObject(unsigned int pbo)
  {
    //checkCudaErrors(cudaGLUnregisterBufferObject(pbo));
  }

  void threadSync() { hipDeviceSynchronize(); }

  void
    calculateNForces(float* Pos, float* Force, float* host_pressure, float* host_Pxy,
      int numBodies, float host_L, int Lperiodic, int *host_RDF, float host_dr2, int blockSize, int q)
  {
    int sharedMemSize = blockSize * sizeof(float4);
    int gridSize = (int)((numBodies + blockSize - 1) / blockSize);
    dim3 dimGrid(gridSize);
    dim3 dimBlock(blockSize);
    int calcmaxnumblast = numBodies % blockSize;
    calcmaxnumblast = (calcmaxnumblast == 0) ? blockSize : calcmaxnumblast;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(L),
      &host_L,
      sizeof(float), 0,
      hipMemcpyHostToDevice));

    float host_periodic = static_cast<float>(Lperiodic);

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(periodic),
      &host_periodic,
      sizeof(float), 0,
      hipMemcpyHostToDevice));

    *host_pressure = 0.f;
    *host_Pxy = 0.f;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pressure),
      host_pressure,
      sizeof(float), 0,
      hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Pxy),
      host_Pxy,
      sizeof(float), 0,
      hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dr2),
      &host_dr2,
      sizeof(float), 0,
      hipMemcpyHostToDevice));

    int* RDFall;
    allocateIntArray(&RDFall, 256 * gridSize);

    calculateForces << < dimGrid, dimBlock, sharedMemSize >> >
      ((float4*)Pos, (float4*)Force,
        numBodies, calcmaxnumblast, RDFall);

    checkCudaErrors(hipMemcpyFromSymbol(host_pressure,
      HIP_SYMBOL(pressure),
      sizeof(float), 0,
      hipMemcpyDeviceToHost));

    checkCudaErrors(hipMemcpyFromSymbol(host_Pxy,
      HIP_SYMBOL(Pxy),
      sizeof(float), 0,
      hipMemcpyDeviceToHost));

    int* RDFtot;
    allocateIntArray(&RDFtot, 256);

    RDFmerger << <   dimGrid, dimBlock, sharedMemSize >> >
      (RDFall, gridSize, RDFtot);

    checkCudaErrors(hipMemcpy(host_RDF, RDFtot, 256 * sizeof(int),
      hipMemcpyDeviceToHost));

    deleteIntArray(RDFall);
    deleteIntArray(RDFtot);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
  }


  void threadExit()
  {
    hipDeviceReset();
  }

}
